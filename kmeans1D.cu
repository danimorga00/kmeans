
#include <hip/hip_runtime.h>
#include <time.h>
#include <cstdio>
#include <chrono>
#include <iostream>
#include <vector>
#include <fstream>

#define TPB 512
#define MAX_ITER 50

__device__ float distance(float x1, float x2)
{
	return sqrt((x2-x1)*(x2-x1));
}

__global__ void kMeansClusterAssignment(int *d_datapoints, int *d_clust_assn, int *d_centroids, int N, int K)
{
	//get idx for this datapoint
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//find the closest centroid to this datapoint
	float min_dist = INFINITY;
	int closest_centroid = 0;

	for(int c = 0; c<K;c++)
	{
		float dist = distance(d_datapoints[idx],d_centroids[c]);

		if(dist < min_dist)
		{
			min_dist = dist;
			closest_centroid=c;
		}
	}

	//assign closest cluster id for this datapoint/thread
	d_clust_assn[idx]=closest_centroid;
	__syncthreads();
}
__global__ void accumulateCentroid(int *d_datapoints, int *d_clust_assn,
                                     int *d_centroids, int *d_clust_sizes, int N, int K) {
    // Indice globale del thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Controlliamo che il thread stia lavorando su un dato valido
    if (tid < N) {
        // Ottieni il punto corrente
        int point = d_datapoints[tid];

        // Ottieni l'ID del centroide a cui è stato assegnato questo punto
        int clusterId = d_clust_assn[tid];

        // Aggiorna il centroide corrispondente e il numero di punti associati (in modo atomico)
        atomicAdd(&(d_centroids[clusterId]), point);
        atomicAdd(&(d_clust_sizes[clusterId]), 1);
    }
}

__global__ void resetCentroids(int *d_centroids, int *d_clust_sizes, int K) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < K) {
        d_centroids[idx] = 0.0f;
        d_clust_sizes[idx] = 0;
    }
}
__global__ void finalizeCentroids(int *d_centroids, int *d_clust_sizes, int K) {
    // Indice del thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < K) {
        // Evita la divisione per zero
        if (d_clust_sizes[tid] > 0) {
            d_centroids[tid] /= d_clust_sizes[tid];
        } else {
            // In caso di cluster vuoto, si può lasciare il centroide invariato
            // oppure gestire il caso in altro modo (es: randomizzare il centroide)
        }
    }
}
float firstExperiment(int N, int K)
{

	//allocate memory on the device for the data points
	int *d_datapoints=0;
	//allocate memory on the device for the cluster assignments
	int *d_clust_assn = 0;
	//allocate memory on the device for the cluster centroids
	int *d_centroids = 0;
	//allocate memory on the device for the cluster sizes
	int *d_clust_sizes=0;

	hipMalloc(&d_datapoints, N*sizeof(int));
	hipMalloc(&d_clust_assn,N*sizeof(int));
	hipMalloc(&d_centroids,K*sizeof(int));
	hipMalloc(&d_clust_sizes,K*sizeof(int));

	int *h_centroids = (int*)malloc(K*sizeof(int));
	int *h_datapoints = (int*)malloc(N*sizeof(int));
	int *h_clust_assn = (int*)malloc(N*sizeof(int));
	int *h_clust_sizes = (int*)malloc(K*sizeof(int));

	srand(time(0));

	//initialize centroids
	for(int c=0;c<K;++c)
	{
		h_centroids[c]= rand() % 1000;
    	//std::cout << "{" << h_centroids[c]  << "}" << std::endl;
		h_clust_sizes[c]=0;
	}

	//initalize datapoints
	for(int d = 0; d < N; ++d)
	{
		h_datapoints[d] = rand() % 1000;
    //std::cout << "{" << h_datapoints[d]  << "}" << std::endl;
	}

	hipMemcpy(d_centroids,h_centroids,K*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_datapoints,h_datapoints,N*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_clust_sizes,h_clust_sizes,K*sizeof(int),hipMemcpyHostToDevice);

	int cur_iter = 0;

	auto start = std::chrono::high_resolution_clock::now();

	while(cur_iter < MAX_ITER)
	{
		kMeansClusterAssignment<<<(N+TPB-1)/TPB,TPB>>>(d_datapoints,d_clust_assn,d_centroids, N, K);

		hipMemcpy(h_centroids,d_centroids,K*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(h_clust_sizes,d_clust_sizes,K*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(h_clust_assn,d_clust_assn,N*sizeof(int),hipMemcpyDeviceToHost);

		resetCentroids<<<(N+TPB-1)/TPB,TPB>>>(d_centroids, d_clust_sizes,  K);

		accumulateCentroid<<<(N+TPB-1)/TPB,TPB>>>(d_datapoints,d_clust_assn,d_centroids,d_clust_sizes, N, K);

		hipDeviceSynchronize();

		finalizeCentroids<<<(K + TPB - 1) / TPB, TPB>>>(d_centroids, d_clust_sizes, K);

		cur_iter+=1;
	}

	for(int c=0;c<N;++c)
	{
    //std::cout << "{" << h_centroids[c]  << "}" << std::endl;
	}
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  	std::cout << "Tempo di esecuzione: " << ((float)duration.count())/1000 << " millisecondi" << std::endl;

	hipFree(d_datapoints);
	hipFree(d_clust_assn);
	hipFree(d_centroids);
	hipFree(d_clust_sizes);

	free(h_centroids);
	free(h_datapoints);
	free(h_clust_sizes);

	return ((float)duration.count())/1000;
}

typedef struct {
	float time;
	int numPoints;
	int numClusters;
	int tpb;
}ExperimentResult;

// Funzione per scrivere il vettore di struct in un file CSV
void writeToCSV(const std::vector<ExperimentResult>& results, const std::string& filename) {
	// Apri il file in modalità scrittura
	std::ofstream file;
	file.open (filename);
	// Controlla se il file è aperto correttamente
	if (!file.is_open()) {
		std::cerr << "Errore nell'aprire il file!" << std::endl;
		return;
	}

	// Scrivi l'intestazione del CSV (opzionale)
	file << "numPoints,numClusters,tpb, time\n";

	// Itera attraverso la lista di risultati e scrivi ogni struct nel CSV
	for (const auto& result : results) {
		file << result.numPoints << "," << result.numClusters << "," << result.tpb << "," << result.time << "\n";
	}

	// Chiudi il file
	file.close();

	std::cout << "File CSV scritto correttamente." << std::endl;
}

int main(){
	int it = 10;
	int j=0;

	std::vector<ExperimentResult> results;
	ExperimentResult result = {0};

	for(int i=0;i<it; i++){
		j=pow(2,i);
  		result.time = firstExperiment(500000, 10*j);
  		result.numPoints = 500000;
  		result.numClusters = 10*j;
  		result.tpb = TPB;
        results.push_back(result);
  	}
    writeToCSV(results, "exp2_par.csv");     //FIXME non crea il file
}