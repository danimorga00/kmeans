
#include <hip/hip_runtime.h>
#include <time.h>
#include <cstdio>
#include <chrono>
#include <iostream>

#define TPB 64
#define MAX_ITER 5

__device__ float distance(float x1, float x2)
{
	return sqrt((x2-x1)*(x2-x1));
}

__global__ void kMeansClusterAssignment(float *d_datapoints, int *d_clust_assn, float *d_centroids, int N, int K)
{
	//get idx for this datapoint
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//find the closest centroid to this datapoint
	float min_dist = INFINITY;
	int closest_centroid = 0;

	for(int c = 0; c<K;c++)
	{
		float dist = distance(d_datapoints[idx],d_centroids[c]);

		if(dist < min_dist)
		{
			min_dist = dist;
			closest_centroid=c;
		}
	}

	//assign closest cluster id for this datapoint/thread
	d_clust_assn[idx]=closest_centroid;
	__syncthreads();
}


__global__ void kMeansCentroidUpdate(float *d_datapoints, int *d_clust_assn, float *d_centroids, int *d_clust_sizes, int N, int K)
{

	//get idx of thread at grid level
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//bounds check
	if (idx >= N) return;

	//get idx of thread at the block level
	const int s_idx = threadIdx.x;

	//put the datapoints and corresponding cluster assignments in shared memory so that they can be summed by thread 0 later
	__shared__ float s_datapoints[TPB];
	s_datapoints[s_idx]= d_datapoints[idx];

	__shared__ int s_clust_assn[TPB];
	s_clust_assn[s_idx] = d_clust_assn[idx];

	__syncthreads();

	//it is the thread with idx 0 (in each block) that sums up all the values within the shared array for the block it is in
	if(s_idx==0)
	{
		float* b_clust_datapoint_sums=(float*)malloc(K*sizeof(float));
		int* b_clust_sizes=(int*)malloc(K*sizeof(float));

		for(int j=0; j< blockDim.x; j++)
		{
			int clust_id = s_clust_assn[j];
			b_clust_datapoint_sums[clust_id]+=s_datapoints[j];
			b_clust_sizes[clust_id]+=1;
		}

		//Now we add the sums to the global centroids and add the counts to the global counts.
		for(int z=0; z < K; z++)
		{
			atomicAdd(&d_centroids[z],b_clust_datapoint_sums[z]);
			atomicAdd(&d_clust_sizes[z],b_clust_sizes[z]);
		}
	}

	__syncthreads();

	//currently centroids are just sums, so divide by size to get actual centroids
	if(idx < K){
		d_centroids[idx] = d_centroids[idx]/d_clust_sizes[idx];
	}
	__syncthreads();
}


int firstExperiment(int N, int K)
{

	//allocate memory on the device for the data points
	float *d_datapoints=0;
	//allocate memory on the device for the cluster assignments
	int *d_clust_assn = 0;
	//allocate memory on the device for the cluster centroids
	float *d_centroids = 0;
	//allocate memory on the device for the cluster sizes
	int *d_clust_sizes=0;

	hipMalloc(&d_datapoints, N*sizeof(float));
	hipMalloc(&d_clust_assn,N*sizeof(int));
	hipMalloc(&d_centroids,K*sizeof(float));
	hipMalloc(&d_clust_sizes,K*sizeof(float));

	float *h_centroids = (float*)malloc(K*sizeof(float));
	float *h_datapoints = (float*)malloc(N*sizeof(float));
	int *h_clust_assn = (int*)malloc(N*sizeof(int));
	int *h_clust_sizes = (int*)malloc(K*sizeof(int));

	srand(time(0));

	//initialize centroids
	for(int c=0;c<K;++c)
	{
		h_centroids[c]=(float) rand() / (double)RAND_MAX;
    std::cout << "{" << h_centroids[c]  << "}" << std::endl;
		h_clust_sizes[c]=0;
	}

	//initalize datapoints
	for(int d = 0; d < N; ++d)
	{
		h_datapoints[d] = (float) rand() / (double)RAND_MAX;
    //std::cout << "{" << h_datapoints[d]  << "}" << std::endl;
	}

	hipMemcpy(d_centroids,h_centroids,K*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_datapoints,h_datapoints,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_clust_sizes,h_clust_sizes,K*sizeof(int),hipMemcpyHostToDevice);

	int cur_iter = 1;

	auto start = std::chrono::high_resolution_clock::now();

	while(cur_iter < MAX_ITER)
	{
		//call cluster assignment kernel
		kMeansClusterAssignment<<<(N+TPB-1)/TPB,TPB>>>(d_datapoints,d_clust_assn,d_centroids, N, K);

		//copy new centroids back to host
		hipMemcpy(h_centroids,d_centroids,K*sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(h_clust_sizes,d_clust_sizes,K*sizeof(int),hipMemcpyDeviceToHost);

		hipMemcpy(h_clust_assn,d_clust_assn,N*sizeof(int),hipMemcpyDeviceToHost);
		printf("Iteration %d: point 0: %f --> %d\n",cur_iter,h_datapoints[0],h_clust_assn[0]);

		//reset centroids and cluster sizes (will be updated in the next kernel)
		hipMemset(d_centroids,0.0,K*sizeof(float));
		hipMemset(d_clust_sizes,0,K*sizeof(int));

		//call centroid update kernel
		kMeansCentroidUpdate<<<(N+TPB-1)/TPB,TPB>>>(d_datapoints,d_clust_assn,d_centroids,d_clust_sizes, N, K);

		for(int i =0; i < K; ++i){
			printf("Iteration %d: centroid %d: %f, cluster size: %d\n",cur_iter,i,h_centroids[i], h_clust_sizes[i]);
		}

		cur_iter+=1;
	}

	for(int c=0;c<K;++c)
	{
    //std::cout << "{" << h_centroids[c]  << "}" << std::endl;
	}
	auto end = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
  std::cout << "Tempo di esecuzione: " << ((float)duration.count())/1000 << " millisecondi" << std::endl;

	hipFree(d_datapoints);
	hipFree(d_clust_assn);
	hipFree(d_centroids);
	hipFree(d_clust_sizes);

	free(h_centroids);
	free(h_datapoints);
	free(h_clust_sizes);

	return 0;
}

int main(){
  firstExperiment(100000, 3);
}